#include"hip/hip_runtime.h"

#include"hiprand/hiprand_kernel.h"// this lib shoulb be included
#include<ctime>
#include<iostream>
#include<random>

using namespace std;

__constant__ int versions[6][3]={{64,64,32},{64,128,36},{64,192,40},{128,128,40},{128,256,48},{128,384,56}};

__constant__ int N2=1024*8;
int hN2=1024*8;

/***test for skinny-128-384***/
/*int hN1=16;
__constant__ int N1=16;
__constant__ int ver=5;
__constant__ int r=3;
__constant__ unsigned char dk1[48] = {0,0,0,0, 0,0,0,0x24, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0xc6, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0x83, 0,0,0,0, 0,0,0,0};
__constant__ unsigned char dk2[48] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0x8a,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0x87,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0x33,0,0};
__constant__ unsigned char dp[16] = {0,4,0,4, 4,0,4,0, 0,4,0,0, 0,0,4,0};
__constant__ unsigned char dc[16] = {0,0,0,0, 0,2,0,66, 2,0,0,0, 0,0,0,66};*/


/***test for skinny-64-128***/
/*int hN1=64;
__constant__ int N1=64;
__constant__ int ver=1;
__constant__ int r=6;
__constant__ unsigned char dk1[24] = {00,0x04, 00,00, 00,00, 00,0x0a, 00,0x03, 00,00, 00,00, 00,0x05,};
__constant__ unsigned char dk2[24] = {00,00, 00,00, 00,00, 0x04,00, 00,00, 00,00, 00,00, 0x0c,00,};
__constant__ unsigned char dp[8] = {0xf0,00, 00,00, 00,00, 0xf0,00}; 
__constant__ unsigned char dc[8] = {00,00, 00,00, 00,00, 00,00};*/


/***test for skinny-128-256***/
//19r
int hN1=256;
__constant__ int N1=256;
__constant__ int ver=4;
__constant__ int r=4;
__constant__ unsigned char dk1[48] = {0,0,0,0, 0,0,0,0x6f, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0x7e, 0,0,0,0, 0,0,0,0,};
__constant__ unsigned char dk2[48] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0x80,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0xa8,0,0,};
__constant__ unsigned char dp[16] = {0,4,0,4, 4,0,4,0, 0,4,0,0, 0,0,4,0};
__constant__ unsigned char dc[16] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,32,0};
//18r
/*int hN1=8192;
__constant__ int N1=8192;
__constant__ int ver=4;
__constant__ int r=4;
__constant__ unsigned char dk1[48] = {0,0,0,0, 0,0xf0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0xc0,0,0, 0,0,0,0, 0,0,0,0,};
__constant__ unsigned char dk2[48] = {0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0xed,0, 0,0,0,0, 0,0,0,0, 0,0,0,0, 0,0,0xb7,0,};
__constant__ unsigned char dp[16] = {0,64,0,0, 0,0,0,0, 0,64,0,0, 0,64,0,0};
__constant__ unsigned char dc[16] = {0,128,0,16, 48,0,0,48, 0,0,0,160, 48,0,0,0};*/

/***test for skinny-64-192***/
/*int hN1=8;
__constant__ int N1=8;
__constant__ int ver=2;
__constant__ int r=6;
__constant__ unsigned char dk1[24] = {00,0x0f, 00,0xa0, 00,00, 0x00,0x50, 00,0x0f, 00,0xa0, 00,00, 00,0x50, 00,0x03, 00,0x10, 00,00, 00,0x20};
__constant__ unsigned char dk2[24] = {00,00, 00,00, 00,00, 0x01,00, 00,00, 00,00, 00,00, 0x0a,00, 00,00, 00,00, 00,00, 0x02,00,};
__constant__ unsigned char dp[8] = {00,0x02, 00,00, 00,00, 00,0x02}; 
__constant__ unsigned char dc[8] = {00,00, 00,00, 00,00, 00,00};*/


// 4-bit Sbox
__constant__ unsigned char sbox_4[16] = {12,6,9,0,1,10,2,11,3,8,5,13,4,14,7,15};
__constant__ unsigned char sbox_4_inv[16] = {3,4,6,8,12,10,1,14,9,2,5,7,0,11,13,15};

// 8-bit Sbox
__constant__ unsigned char sbox_8[256] = {0x65 , 0x4c , 0x6a , 0x42 , 0x4b , 0x63 , 0x43 , 0x6b , 0x55 , 0x75 , 0x5a , 0x7a , 0x53 , 0x73 , 0x5b , 0x7b ,0x35 , 0x8c , 0x3a , 0x81 , 0x89 , 0x33 , 0x80 , 0x3b , 0x95 , 0x25 , 0x98 , 0x2a , 0x90 , 0x23 , 0x99 , 0x2b ,0xe5 , 0xcc , 0xe8 , 0xc1 , 0xc9 , 0xe0 , 0xc0 , 0xe9 , 0xd5 , 0xf5 , 0xd8 , 0xf8 , 0xd0 , 0xf0 , 0xd9 , 0xf9 ,0xa5 , 0x1c , 0xa8 , 0x12 , 0x1b , 0xa0 , 0x13 , 0xa9 , 0x05 , 0xb5 , 0x0a , 0xb8 , 0x03 , 0xb0 , 0x0b , 0xb9 ,0x32 , 0x88 , 0x3c , 0x85 , 0x8d , 0x34 , 0x84 , 0x3d , 0x91 , 0x22 , 0x9c , 0x2c , 0x94 , 0x24 , 0x9d , 0x2d ,0x62 , 0x4a , 0x6c , 0x45 , 0x4d , 0x64 , 0x44 , 0x6d , 0x52 , 0x72 , 0x5c , 0x7c , 0x54 , 0x74 , 0x5d , 0x7d ,0xa1 , 0x1a , 0xac , 0x15 , 0x1d , 0xa4 , 0x14 , 0xad , 0x02 , 0xb1 , 0x0c , 0xbc , 0x04 , 0xb4 , 0x0d , 0xbd ,0xe1 , 0xc8 , 0xec , 0xc5 , 0xcd , 0xe4 , 0xc4 , 0xed , 0xd1 , 0xf1 , 0xdc , 0xfc , 0xd4 , 0xf4 , 0xdd , 0xfd ,0x36 , 0x8e , 0x38 , 0x82 , 0x8b , 0x30 , 0x83 , 0x39 , 0x96 , 0x26 , 0x9a , 0x28 , 0x93 , 0x20 , 0x9b , 0x29 ,0x66 , 0x4e , 0x68 , 0x41 , 0x49 , 0x60 , 0x40 , 0x69 , 0x56 , 0x76 , 0x58 , 0x78 , 0x50 , 0x70 , 0x59 , 0x79 ,0xa6 , 0x1e , 0xaa , 0x11 , 0x19 , 0xa3 , 0x10 , 0xab , 0x06 , 0xb6 , 0x08 , 0xba , 0x00 , 0xb3 , 0x09 , 0xbb ,0xe6 , 0xce , 0xea , 0xc2 , 0xcb , 0xe3 , 0xc3 , 0xeb , 0xd6 , 0xf6 , 0xda , 0xfa , 0xd3 , 0xf3 , 0xdb , 0xfb ,0x31 , 0x8a , 0x3e , 0x86 , 0x8f , 0x37 , 0x87 , 0x3f , 0x92 , 0x21 , 0x9e , 0x2e , 0x97 , 0x27 , 0x9f , 0x2f ,0x61 , 0x48 , 0x6e , 0x46 , 0x4f , 0x67 , 0x47 , 0x6f , 0x51 , 0x71 , 0x5e , 0x7e , 0x57 , 0x77 , 0x5f , 0x7f ,0xa2 , 0x18 , 0xae , 0x16 , 0x1f , 0xa7 , 0x17 , 0xaf , 0x01 , 0xb2 , 0x0e , 0xbe , 0x07 , 0xb7 , 0x0f , 0xbf ,0xe2 , 0xca , 0xee , 0xc6 , 0xcf , 0xe7 , 0xc7 , 0xef , 0xd2 , 0xf2 , 0xde , 0xfe , 0xd7 , 0xf7 , 0xdf , 0xff};
__constant__ unsigned char sbox_8_inv[256] = {0xac , 0xe8 , 0x68 , 0x3c , 0x6c , 0x38 , 0xa8 , 0xec , 0xaa , 0xae , 0x3a , 0x3e , 0x6a , 0x6e , 0xea , 0xee ,0xa6 , 0xa3 , 0x33 , 0x36 , 0x66 , 0x63 , 0xe3 , 0xe6 , 0xe1 , 0xa4 , 0x61 , 0x34 , 0x31 , 0x64 , 0xa1 , 0xe4 ,0x8d , 0xc9 , 0x49 , 0x1d , 0x4d , 0x19 , 0x89 , 0xcd , 0x8b , 0x8f , 0x1b , 0x1f , 0x4b , 0x4f , 0xcb , 0xcf ,0x85 , 0xc0 , 0x40 , 0x15 , 0x45 , 0x10 , 0x80 , 0xc5 , 0x82 , 0x87 , 0x12 , 0x17 , 0x42 , 0x47 , 0xc2 , 0xc7 ,0x96 , 0x93 , 0x03 , 0x06 , 0x56 , 0x53 , 0xd3 , 0xd6 , 0xd1 , 0x94 , 0x51 , 0x04 , 0x01 , 0x54 , 0x91 , 0xd4 ,0x9c , 0xd8 , 0x58 , 0x0c , 0x5c , 0x08 , 0x98 , 0xdc , 0x9a , 0x9e , 0x0a , 0x0e , 0x5a , 0x5e , 0xda , 0xde ,0x95 , 0xd0 , 0x50 , 0x05 , 0x55 , 0x00 , 0x90 , 0xd5 , 0x92 , 0x97 , 0x02 , 0x07 , 0x52 , 0x57 , 0xd2 , 0xd7 ,0x9d , 0xd9 , 0x59 , 0x0d , 0x5d , 0x09 , 0x99 , 0xdd , 0x9b , 0x9f , 0x0b , 0x0f , 0x5b , 0x5f , 0xdb , 0xdf ,0x16 , 0x13 , 0x83 , 0x86 , 0x46 , 0x43 , 0xc3 , 0xc6 , 0x41 , 0x14 , 0xc1 , 0x84 , 0x11 , 0x44 , 0x81 , 0xc4 ,0x1c , 0x48 , 0xc8 , 0x8c , 0x4c , 0x18 , 0x88 , 0xcc , 0x1a , 0x1e , 0x8a , 0x8e , 0x4a , 0x4e , 0xca , 0xce ,0x35 , 0x60 , 0xe0 , 0xa5 , 0x65 , 0x30 , 0xa0 , 0xe5 , 0x32 , 0x37 , 0xa2 , 0xa7 , 0x62 , 0x67 , 0xe2 , 0xe7 ,0x3d , 0x69 , 0xe9 , 0xad , 0x6d , 0x39 , 0xa9 , 0xed , 0x3b , 0x3f , 0xab , 0xaf , 0x6b , 0x6f , 0xeb , 0xef ,0x26 , 0x23 , 0xb3 , 0xb6 , 0x76 , 0x73 , 0xf3 , 0xf6 , 0x71 , 0x24 , 0xf1 , 0xb4 , 0x21 , 0x74 , 0xb1 , 0xf4 ,0x2c , 0x78 , 0xf8 , 0xbc , 0x7c , 0x28 , 0xb8 , 0xfc , 0x2a , 0x2e , 0xba , 0xbe , 0x7a , 0x7e , 0xfa , 0xfe ,0x25 , 0x70 , 0xf0 , 0xb5 , 0x75 , 0x20 , 0xb0 , 0xf5 , 0x22 , 0x27 , 0xb2 , 0xb7 , 0x72 , 0x77 , 0xf2 , 0xf7 ,0x2d , 0x79 , 0xf9 , 0xbd , 0x7d , 0x29 , 0xb9 , 0xfd , 0x2b , 0x2f , 0xbb , 0xbf , 0x7b , 0x7f , 0xfb , 0xff};

// ShiftAndSwitchRows permutation
__constant__ unsigned char P[16] = {0,1,2,3,7,4,5,6,10,11,8,9,13,14,15,12};
__constant__ unsigned char P_inv[16] = {0,1,2,3,5,6,7,4,10,11,8,9,15,12,13,14};

// Tweakey permutation
__constant__ unsigned char TWEAKEY_P[16] = {9,15,8,13,10,14,12,11,0,1,2,3,4,5,6,7};
__constant__ unsigned char TWEAKEY_P_inv[16] = {8,9,10,11,12,13,14,15,2,0,4,7,6,3,5,1};

// round constants
__constant__ unsigned char RC[62] = {
		0x01, 0x03, 0x07, 0x0F, 0x1F, 0x3E, 0x3D, 0x3B, 0x37, 0x2F,
		0x1E, 0x3C, 0x39, 0x33, 0x27, 0x0E, 0x1D, 0x3A, 0x35, 0x2B,
		0x16, 0x2C, 0x18, 0x30, 0x21, 0x02, 0x05, 0x0B, 0x17, 0x2E,
		0x1C, 0x38, 0x31, 0x23, 0x06, 0x0D, 0x1B, 0x36, 0x2D, 0x1A,
		0x34, 0x29, 0x12, 0x24, 0x08, 0x11, 0x22, 0x04, 0x09, 0x13,
		0x26, 0x0c, 0x19, 0x32, 0x25, 0x0a, 0x15, 0x2a, 0x14, 0x28,
		0x10, 0x20};

__device__ void AddKey(unsigned char state[4][4], unsigned char keyCells[3][4][4], int ver)
{
	int i, j, k;
	unsigned char pos;
	unsigned char keyCells_tmp[3][4][4];


    for(i = 0; i <= 1; i++)
    {
        for(j = 0; j < 4; j++)
        {
            state[i][j] ^= keyCells[0][i][j];
            if (2*versions[ver][0]==versions[ver][1])
                state[i][j] ^= keyCells[1][i][j];
            else if (3*versions[ver][0]==versions[ver][1])
               state[i][j] ^= keyCells[1][i][j] ^ keyCells[2][i][j];
        }
    }


    for(k = 0; k <(int)(versions[ver][1]/versions[ver][0]); k++){
        for(i = 0; i < 4; i++){
            for(j = 0; j < 4; j++){
        
                pos=TWEAKEY_P[j+4*i];
                keyCells_tmp[k][i][j]=keyCells[k][pos>>2][pos&0x3];
            }
        }
    }

    for(k = 0; k <(int)(versions[ver][1]/versions[ver][0]); k++){
        for(i = 0; i <= 1; i++){
            for(j = 0; j < 4; j++){
                if (k==1)
                {
                    if (versions[ver][0]==64)
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]<<1)&0xE)^((keyCells_tmp[k][i][j]>>3)&0x1)^((keyCells_tmp[k][i][j]>>2)&0x1);
                    else
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]<<1)&0xFE)^((keyCells_tmp[k][i][j]>>7)&0x01)^((keyCells_tmp[k][i][j]>>5)&0x01);
                }
                else if (k==2)
                {
                    if (versions[ver][0]==64)
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]>>1)&0x7)^((keyCells_tmp[k][i][j])&0x8)^((keyCells_tmp[k][i][j]<<3)&0x8);
                    else
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]>>1)&0x7F)^((keyCells_tmp[k][i][j]<<7)&0x80)^((keyCells_tmp[k][i][j]<<1)&0x80);
                }
            }
        }
    }

    for(k = 0; k <(int)(versions[ver][1]/versions[ver][0]); k++){
        for(i = 0; i < 4; i++){
            for(j = 0; j < 4; j++){
                keyCells[k][i][j]=keyCells_tmp[k][i][j];
            }
        }
    }
}


__device__ void AddKey_inv(unsigned char state[4][4], unsigned char keyCells[3][4][4], int ver)
{
	int i, j, k;
	unsigned char pos;
	unsigned char keyCells_tmp[3][4][4];
    for(k = 0; k <(int)(versions[ver][1]/versions[ver][0]); k++){
        for(i = 0; i < 4; i++){
            for(j = 0; j < 4; j++){
                
                pos=TWEAKEY_P_inv[j+4*i];
                keyCells_tmp[k][i][j]=keyCells[k][pos>>2][pos&0x3];
            }
        }
    }

    for(k = 0; k <(int)(versions[ver][1]/versions[ver][0]); k++){
        for(i = 2; i <= 3; i++){
            for(j = 0; j < 4; j++){
           
                if (k==1)
                {
                    if (versions[ver][0]==64)
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]>>1)&0x7)^((keyCells_tmp[k][i][j]<<3)&0x8)^((keyCells_tmp[k][i][j])&0x8);
                    else
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]>>1)&0x7F)^((keyCells_tmp[k][i][j]<<7)&0x80)^((keyCells_tmp[k][i][j]<<1)&0x80);
                }
                else if (k==2)
                {
                    if (versions[ver][0]==64)
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]<<1)&0xE)^((keyCells_tmp[k][i][j]>>3)&0x1)^((keyCells_tmp[k][i][j]>>2)&0x1);
                    else
                        keyCells_tmp[k][i][j]=((keyCells_tmp[k][i][j]<<1)&0xFE)^((keyCells_tmp[k][i][j]>>7)&0x01)^((keyCells_tmp[k][i][j]>>5)&0x01);
                }
            }
        }
    }

    for(k = 0; k <(int)(versions[ver][1]/versions[ver][0]); k++){
        for(i = 0; i < 4; i++){
            for(j = 0; j < 4; j++){
                keyCells[k][i][j]=keyCells_tmp[k][i][j];
            }
        }
    }

    for(i = 0; i <= 1; i++)
    {
        for(j = 0; j < 4; j++)
        {
            state[i][j] ^= keyCells[0][i][j];
            if (2*versions[ver][0]==versions[ver][1])
                state[i][j] ^= keyCells[1][i][j];
            else if (3*versions[ver][0]==versions[ver][1])
                state[i][j] ^= keyCells[1][i][j] ^ keyCells[2][i][j];
        }
    }
}


__device__ void AddConstants(unsigned char state[4][4], int r)
{
	state[0][0] ^= (RC[r] & 0xf);
	state[1][0] ^= ((RC[r]>>4) & 0x3);
	state[2][0] ^= 0x2;
}

__device__ void SubCell4(unsigned char state[4][4])
{
	int i,j;
	for(i = 0; i < 4; i++)
		for(j = 0; j <  4; j++)
			state[i][j] = sbox_4[state[i][j]];
}


__device__ void SubCell4_inv(unsigned char state[4][4])
{
	int i,j;
	for(i = 0; i < 4; i++)
		for(j = 0; j <  4; j++)
			state[i][j] = sbox_4_inv[state[i][j]];
}


__device__ void SubCell8(unsigned char state[4][4])
{
	int i,j;
	for(i = 0; i < 4; i++)
		for(j = 0; j <  4; j++)
			state[i][j] = sbox_8[state[i][j]];
}


__device__ void SubCell8_inv(unsigned char state[4][4])
{
	int i,j;
	for(i = 0; i < 4; i++)
		for(j = 0; j <  4; j++)
			state[i][j] = sbox_8_inv[state[i][j]];
}


__device__ void ShiftRows(unsigned char state[4][4])
{
	int i, j, pos;

	unsigned char state_tmp[4][4];
    for(i = 0; i < 4; i++)
    {
        for(j = 0; j < 4; j++)
        {
            pos=P[j+4*i];
            state_tmp[i][j]=state[pos>>2][pos&0x3];
        }
    }

    for(i = 0; i < 4; i++)
    {
        for(j = 0; j < 4; j++)
        {
            state[i][j]=state_tmp[i][j];
        }
    }
}

__device__ void ShiftRows_inv(unsigned char state[4][4])
{
	int i, j, pos;

	unsigned char state_tmp[4][4];
    for(i = 0; i < 4; i++)
    {
        for(j = 0; j < 4; j++)
        {
            pos=P_inv[j+4*i];
            state_tmp[i][j]=state[pos>>2][pos&0x3];
        }
    }

    for(i = 0; i < 4; i++)
    {
        for(j = 0; j < 4; j++)
        {
            state[i][j]=state_tmp[i][j];
        }
    }
}


__device__ void MixColumn(unsigned char state[4][4])
{
	int j;
    unsigned char temp;

	for(j = 0; j < 4; j++){
        state[1][j]^=state[2][j];
        state[2][j]^=state[0][j];
        state[3][j]^=state[2][j];

        temp=state[3][j];
        state[3][j]=state[2][j];
        state[2][j]=state[1][j];
        state[1][j]=state[0][j];
        state[0][j]=temp;
	}
}


__device__ void MixColumn_inv(unsigned char state[4][4])
{
	int j;
    unsigned char temp;

	for(j = 0; j < 4; j++){
        temp=state[3][j];
        state[3][j]=state[0][j];
        state[0][j]=state[1][j];
        state[1][j]=state[2][j];
        state[2][j]=temp;

        state[3][j]^=state[2][j];
        state[2][j]^=state[0][j];
        state[1][j]^=state[2][j];
	}
}

__device__ void ENC(unsigned char* input, const unsigned char* userkey, int ver, int r)
{
	unsigned char state[4][4];
	unsigned char keyCells[3][4][4];
	int i;

	for(i=0; i<3; i++)
		for(int j=0; j<4; j++)
			for(int k=0; k<4; k++)
				keyCells[i][j][k]=0;                                 
	for(i = 0; i < 16; i++) {
        if (versions[ver][0]==64){
            if(i&1)
            {
                state[i>>2][i&0x3] = input[i>>1]&0xF;
                keyCells[0][i>>2][i&0x3] = userkey[i>>1]&0xF;
                if (versions[ver][1]>=128)
                    keyCells[1][i>>2][i&0x3] = userkey[(i+16)>>1]&0xF;
                if (versions[ver][1]>=192)
                    keyCells[2][i>>2][i&0x3] = userkey[(i+32)>>1]&0xF;
            }
            else
            {
                state[i>>2][i&0x3] = (input[i>>1]>>4)&0xF;
                keyCells[0][i>>2][i&0x3] = (userkey[i>>1]>>4)&0xF;
                if (versions[ver][1]>=128)
                    keyCells[1][i>>2][i&0x3] = (userkey[(i+16)>>1]>>4)&0xF;
                if (versions[ver][1]>=192)
                    keyCells[2][i>>2][i&0x3] = (userkey[(i+32)>>1]>>4)&0xF;
            }
        }
        else if (versions[ver][0]==128){
            state[i>>2][i&0x3] = input[i]&0xFF;
            keyCells[0][i>>2][i&0x3] = userkey[i]&0xFF;
            if (versions[ver][1]>=256)
                keyCells[1][i>>2][i&0x3] = userkey[i+16]&0xFF;
            if (versions[ver][1]>=384)
                keyCells[2][i>>2][i&0x3] = userkey[i+32]&0xFF;
        }
    }


	for(i = 0; i < r; i++){
        if (versions[ver][0]==64)
            SubCell4(state);
        else
            SubCell8(state);
 
        AddConstants(state, i);
           
        AddKey(state, keyCells, ver);
         
        ShiftRows(state);
            
        MixColumn(state);
           
		
	}

	

    if (versions[ver][0]==64)
    {
        for(i = 0; i < 8; i++)
            input[i] = ((state[(2*i)>>2][(2*i)&0x3] & 0xF) << 4) | (state[(2*i+1)>>2][(2*i+1)&0x3] & 0xF);
    }
    else if (versions[ver][0]==128)
    {
        for(i = 0; i < 16; i++)
            input[i] = state[i>>2][i&0x3] & 0xFF;
    }

}


__device__ void DEC(unsigned char* input, const unsigned char* userkey, int ver, int r)
{
	unsigned char state[4][4];
	unsigned char dummy[4][4]={{0}};
	unsigned char keyCells[3][4][4];
	int i;

    	for(i=0; i<3; i++)
		for(int j=0; j<4; j++)
			for(int k=0; k<4; k++)
				keyCells[i][j][k]=0;
	for(i = 0; i < 16; i++) {
        if (versions[ver][0]==64){
            if(i&1)
            {
                state[i>>2][i&0x3] = input[i>>1]&0xF;
                keyCells[0][i>>2][i&0x3] = userkey[i>>1]&0xF;
                if (versions[ver][1]>=128)
                    keyCells[1][i>>2][i&0x3] = userkey[(i+16)>>1]&0xF;
                if (versions[ver][1]>=192)
                    keyCells[2][i>>2][i&0x3] = userkey[(i+32)>>1]&0xF;
            }
            else
            {
                state[i>>2][i&0x3] = (input[i>>1]>>4)&0xF;
                keyCells[0][i>>2][i&0x3] = (userkey[i>>1]>>4)&0xF;
                if (versions[ver][1]>=128)
                    keyCells[1][i>>2][i&0x3] = (userkey[(i+16)>>1]>>4)&0xF;
                if (versions[ver][1]>=192)
                    keyCells[2][i>>2][i&0x3] = (userkey[(i+32)>>1]>>4)&0xF;
            }
        }
        else if (versions[ver][0]==128){
            state[i>>2][i&0x3] = input[i]&0xFF;

            keyCells[0][i>>2][i&0x3] = userkey[i]&0xFF;
            if (versions[ver][1]>=256)
                keyCells[1][i>>2][i&0x3] = userkey[i+16]&0xFF;
            if (versions[ver][1]>=384)
                keyCells[2][i>>2][i&0x3] = userkey[i+32]&0xFF;
        }
    }

    for(i = r-1; i >=0 ; i--){
        AddKey(dummy, keyCells, ver);
    }

    

	for(i = r-1; i >=0 ; i--){
        MixColumn_inv(state);
            
        ShiftRows_inv(state);
           
        AddKey_inv(state, keyCells, ver);
           
        AddConstants(state, i);
            
        if (versions[ver][0]==64)
            SubCell4_inv(state);
        else
            SubCell8_inv(state);
            
	}

	

    if (versions[ver][0]==64)
    {
        for(i = 0; i < 8; i++)
            input[i] = ((state[(2*i)>>2][(2*i)&0x3] & 0xF) << 4) | (state[(2*i+1)>>2][(2*i+1)&0x3] & 0xF);
    }
    else if (versions[ver][0]==128)
    {
        for(i = 0; i < 16; i++)
            input[i] = state[i>>2][i&0x3] & 0xFF;
    }
}

//-------------------generate random numbers-------//
__device__ float generate(hiprandState *globalState, int ind)
{
	hiprandState localState = globalState[ind];
	float RANDOM = hiprand_uniform(&localState);// uniform distribution
	globalState[ind] = localState;
	return RANDOM;
}

__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = iy * blockDim.x*gridDim.x + ix;
	
	hiprand_init(seed, idx, 0, &state[idx]);// initialize the state
}

//-------------This is our kernel function where the random numbers generated------//
__global__ void our_kernel(hiprandState *globalState,int *devNum)
{
	int ix = threadIdx.x + blockIdx.x*blockDim.x;
	int iy = threadIdx.y + blockIdx.y*blockDim.y;
	int idx = iy * blockDim.x*gridDim.x + ix;

	int i,j,jj;
	int num=0;
	unsigned char p1[16],p2[16];
	unsigned char c3[16],c4[16];
	unsigned char k1[48], k2[48], k3[48], k4[48];
	bool flag;
	
	int k;
	// randomly choose k1
	for(i = 0; i < (versions[ver][1]>>3); i++) 
	{
		k = generate(globalState, idx) * 100000;
		k1[i] = k & 0xff;
		//printf("k1[%d]=%d\n", i,k1[i]);
	}	
	for(i = 0; i < (versions[ver][1]>>3); i++) 
		k2[i] = k1[i]^dk1[i];	
	for(i = 0; i < (versions[ver][1]>>3); i++) 
		k3[i] = k1[i]^dk2[i];	
	for(i = 0; i < (versions[ver][1]>>3); i++) 
		k4[i] = k2[i]^dk2[i];
	
	for (j=0; j<N1; j++)
	{
	for (jj=0; jj<N2; jj++)
	{
		//printf("[%d]=%d\n",idx,j);
		// randomly choose p1
		for(i = 0; i < (versions[ver][0]>>3); i++) 
		{
			k = generate(globalState, idx) * 100000;
			p1[i] = k & 0xff;
			//printf("P1[%d]=%d\n", i,p1[i]);	
		}
		// derive p2
		for(i = 0; i < (versions[ver][0]>>3); i++) 
			p2[i] = p1[i]^dp[i];	

		ENC(p1, k1, ver, r);
		ENC(p2, k2, ver, r);
		
		// derive c3
		for(i = 0; i < (versions[ver][0]>>3); i++) 
			c3[i] = p1[i]^dc[i];
		// derive c4
		for(i = 0; i < (versions[ver][0]>>3); i++) 
			c4[i] = p2[i]^dc[i];
		DEC(c3, k3, ver, r);
		DEC(c4, k4, ver, r);
		flag = 1;
		for(i = 0; i < (versions[ver][0]>>3); i++)
		{
			//printf("c3=%d, c4=%d\n", c3[i],c4[i]);	
			if ((c3[i]^c4[i]) != dp[i])
				flag = 0;
		}
		if (flag) 
		{
			num ++; 
		}		
		//printf("%d\n", k);
	}
	}
	
	devNum[idx]=num;
	//printf("%d\n",devNum[idx]);
}

int main()
{
	int blockx = 512;
	int blocky = 1;
	dim3 block(blockx, blocky);

	int gridx = 1;
	int gridy = 64;
	dim3 grid(gridx,gridy); 

	int N = gridx*gridy*blockx*blocky;// the number of states
	int *hostNum = (int *)malloc(N * sizeof(int));
	double sum=0;

	hipEvent_t start, stop;
	float elapsedTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	//--------------------//
	hiprandState* devStates;
	int* devNum;
	hipError_t err = hipSuccess;
	err=hipMalloc(&devStates, N * sizeof(hiprandState));
	err=hipMalloc((void **)&devNum, N * sizeof(int));
	if(err!=hipSuccess)
    	{
        	printf("the hipMalloc on GPU is failed\n");
        	return 1;
    	}
	printf("SUCCESS\n");

	srand(time(0));
	int seed = rand();

	//  Initialize the states
	setup_kernel <<<grid, block>>> (devStates, seed);

	our_kernel <<<grid, block >>> (devStates,devNum);
 	
	err = hipGetLastError();

    	if (err != hipSuccess)
    	{
        	fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
   	}
	
	err=hipMemcpy(hostNum,devNum,N*sizeof(int),hipMemcpyDeviceToHost);
	
	if (err != hipSuccess)
   	{
        	fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
    	}

	for(int i=0;i<N;i++)
		sum += hostNum[i];
	sum = double(N)/sum;
	//cout <<  log(sum)/log(2.0) << endl;
	cout <<  (-log(sum)/log(2.0)-log(hN1)/log(2.0)-log(hN2)/log(2.0)) << endl;

	hipFree(devNum);
	hipFree(devStates);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << (elapsedTime/1000) <<'s'<< endl; 

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceReset();
	return 0;
}
